#include "hip/hip_runtime.h"

# include "../Core/Core.h"

__global__ void square(float *input, float* output)
{
    uint pos = blockIdx.x;
    output[pos] = input[pos] * input[pos];
}