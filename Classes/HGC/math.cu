#include "hip/hip_runtime.h"

# include "../Core/Core.h"
# include "cuda.prv.h"

__global__ void impl_square(float *input, float* output)
{
    uint pos = threadIdx.x;
    output[pos] = input[pos] * input[pos];
}

NNT_BEGIN_CUDA

void square(float* input, float* output, usize count)
{
    Dimension dim;
    dim->dim = 1;
    dim->count[0] = count;
    
    Parameter in = Parameter::Input(input, NNTValueTypeFloat, count);
    Parameter out = Parameter::Output(output, NNTValueTypeFloat, count);
    
    impl_square<<<dim->dim, dim->count[0]>>>(in, out);
}

NNT_END_CUDA
