
# include "../Core/Core.h"
# include "CudaFunction.h"
# include "cuda.prv.h"

NNT_BEGIN_CXX
NNT_BEGIN_NS(cuda)

Dimension::Dimension()
{
    memset(&ndr, 0, sizeof(ndr));
}

Dimension::~Dimension()
{
    
}

Parameter::Parameter(void* o, void* b, usize sz, bool nb)
: _origin(o), _buffer(b), _sz(sz), _needback(nb)
{
    
}

Parameter::~Parameter()
{
    if (_needback)
        back();
    
    hipFree(_buffer);
}

Parameter Parameter::Input(void* da, NNTValueType vt, usize count)
{
    usize sz = vt_sizeof(vt) * count;
    void* buf;
    hipMalloc(&buf, sz);
    hipMemcpy(buf, da, sz, hipMemcpyHostToDevice);
    return Parameter(da, buf, sz, false);
}

Parameter Parameter::InputOutput(void* da, NNTValueType vt, usize count)
{
    usize sz = vt_sizeof(vt) * count;
    void* buf;
    hipMalloc(&buf, sz);
    hipMemcpy(buf, da, sz, hipMemcpyHostToDevice);
    return Parameter(da, buf, sz, true);
}

Parameter Parameter::Output(void* da, NNTValueType vt, usize count)
{
    usize sz = vt_sizeof(vt) * count;
    void* buf;
    hipMalloc(&buf, sz);
    return Parameter(da, buf, sz, true);
}

void Parameter::back()
{
    hipMemcpy(_origin, _buffer, _sz, hipMemcpyDeviceToHost);
}

NNT_END_NS
NNT_END_CXX
